// vl_nnbboxnms.cu
// brief GPU nms block MEX wrapper
// author Samuel Albanie 

/*
Copyright (C) 2017 Samuel Albanie
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/


#include <matrix.h>
#include <vector>

#include <bits/mexutils.h>
#include <bits/datamex.hpp>
#include <bits/nnbboxnms.hpp>

#if ENABLE_GPU
#include <bits/datacu.hpp>
#endif

#include <assert.h>

/* option codes */
enum {
  opt_verbose = 0,
} ;

/* options */
VLMXOption  options [] = {
  {"Verbose",         0,   opt_verbose          },
  {0,                 0,   0                    }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_BOXES = 0, IN_OVERLAP, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  if (nin < 2) {
    mexErrMsgTxt("There are less than two arguments.") ;
  }

  // backwards mode is not yet supported for nms
  next = 2 ;

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      default: 
        break ;
    }
  }

  vl::MexTensor boxes(context) ;
  boxes.init(in[IN_BOXES]) ;
  boxes.reshape(2) ;
  int box_dims = boxes.getHeight() ;
  float overlap = (float)mxGetScalar(in[IN_OVERLAP]) ;

  if (box_dims != 5) {
    vlmxError(VLMXE_IllegalArgument, "BOXES should have shape 5 x N.") ;
  }

  std::vector<int> output = std::vector<int>(boxes.getWidth()) ; // store nms picks
  int num_kept = 0 ; // track the number kept by nms

  if (verbosity > 0) {
    mexPrintf("vl_nnbboxnms: mode %s; %s\n",  
            (boxes.getDeviceType()==vl::VLDT_GPU)?"gpu":"cpu", "forward") ;
        vl::print("vl_nnbboxnms: boxes: ", boxes) ;
      }
      /* -------------------------------------------------------------- */
      /*                                                    Do the work */
      /* -------------------------------------------------------------- */

      vl::ErrorCode error ;
      error = vl::nnbboxnms_forward(context, 
                                    output, 
                                    boxes, 
                                    overlap,
                                    num_kept) ;

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  out[OUT_RESULT] = mxCreateNumericMatrix(num_kept, 1, mxDOUBLE_CLASS, mxREAL) ;
  double *ptr = mxGetPr(out[OUT_RESULT]) ;
  for (int ii = 0 ; ii < num_kept ; ++ii) 
      ptr[ii] = output[ii] + 1 ;
}
