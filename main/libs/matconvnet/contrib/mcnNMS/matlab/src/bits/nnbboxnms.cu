// file nnbboxnms.cu
// brief nms block
// author Samuel Albanie

/*
Copyright (C) 2017- Samuel Albanie
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnbboxnms.hpp"
#include "impl/bboxnms.hpp"

#if ENABLE_GPU
#include <bits/datacu.hpp>
#endif

#include <cstdio>
#include <assert.h>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                         bboxnms_forward */
/* ---------------------------------------------------------------- */

#define DISPATCH(deviceType,T) \
error = vl::impl::bboxnms<deviceType,T>::forward (context, \
output, \
(T const*) boxes.getMemory(), \
(float) overlap, \
(size_t) boxes.getWidth(),\
num_kept) ;

#define DISPATCH2(deviceType) \
switch (dataType) { \
case VLDT_Float : DISPATCH(deviceType, float) ; \
break ; \
IF_DOUBLE(case VLDT_Double : DISPATCH(deviceType, double) ; \
break ;) \
default: assert(false) ; \
return VLE_Unknown ; \
}

vl::ErrorCode
vl::nnbboxnms_forward(vl::Context& context,
                      std::vector<int> &output,
                      vl::Tensor boxes,
                      float overlap,
                      int &num_kept)
{
  vl::ErrorCode error = VLE_Success ;
  vl::DataType dataType = boxes.getDataType() ;
  
  switch (boxes.getDeviceType())
  {
    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:
      DISPATCH2(vl::VLDT_GPU) ;
    if (error == VLE_Cuda) {
      context.setError(context.getCudaHelper().catchCudaError("GPU")) ;
    }
    break;
#endif

    default:
      assert(false);
      error = vl::VLE_Unknown ;
      break ;
  }
  return context.passError(error, __func__);
}
