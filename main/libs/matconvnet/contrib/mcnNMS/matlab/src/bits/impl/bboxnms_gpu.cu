#include "hip/hip_runtime.h"
// @file bboxnms_gpu.cu
// @brief Bounding Box non maximum supression, heavily based 
// on Shaoqing Ren's Faster R-CNN implementation which 
// can be found here: 
// https://github.com/ShaoqingRen/faster_rcnn/blob/master/functions/nms
// @author Samuel Albanie

/*
Copyright (C) 2017- Samuel Albanie.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bboxnms.hpp"
#include <bits/data.hpp>

#include <assert.h>
#include <float.h>
#include <string.h>
#include <cstdio>
#include <math.h>

#include <algorithm>
#include <vector>
#include <map>

// division + round-up
#define DIVROUNDUP(x,y) ((x)/(y)+((x)%(y)>0))

/* ------------------------------------------------------------ */
/*                                                      kernels */
/* ------------------------------------------------------------ */

enum {
  XMIN = 0, YMIN, XMAX, YMAX,
} ;

// set number of threads per block (should be at least 64)
int const BLOCKSIZE = (sizeof(unsigned long long) * 8) ;

// compute intersection over union on the gpu
template <typename T>
__device__ inline float jaccard(T const * const a, T const * const b)
{
    T left = max(a[XMIN], b[XMIN]) ; 
    T right = min(a[XMAX], b[XMAX]) ;
    T top = max(a[YMIN], b[YMIN]) ;
    T bottom = min(a[YMAX], b[YMAX]) ;
    T width = max(right - left + 1, 0.f) ; 
    T height = max(bottom - top + 1, 0.f) ;
    T intersection = width * height ;
    T aArea = (a[XMAX] - a[XMIN] + 1) * (a[YMAX] - a[YMIN] + 1) ;
    T bArea = (b[XMAX] - b[XMIN] + 1) * (b[YMAX] - b[YMIN] + 1) ;
    return intersection / (aArea + bArea - intersection) ;
}

template <typename T>
__global__ void nmsKernel(const int numBoxes, 
                           const float overlapThresh, 
                           const T *boxes, 
                           unsigned long long *mask, 
                           const int colBlocks)
   {
    const int rowIdx = blockIdx.y ; 
    const int colIdx = blockIdx.x ;
    const int numRows = min(numBoxes - rowIdx * BLOCKSIZE, BLOCKSIZE) ; 
    const int numCols = min(numBoxes - colIdx * BLOCKSIZE, BLOCKSIZE) ;

    // all blocks in the same column of the block-grid will process the same
    // set of boxes
    int offset = BLOCKSIZE * colIdx ;

    // define shared memory for all the boxes processed by the current block
    __shared__ T blockBoxes[BLOCKSIZE * 5] ;

    // load bounding boxes and scores for current block into shared memory
    if (threadIdx.x < numCols)
    {
        blockBoxes[threadIdx.x*5 + 0] = boxes[(offset + threadIdx.x)*5 + 0] ;
        blockBoxes[threadIdx.x*5 + 1] = boxes[(offset + threadIdx.x)*5 + 1] ;
        blockBoxes[threadIdx.x*5 + 2] = boxes[(offset + threadIdx.x)*5 + 2] ;
        blockBoxes[threadIdx.x*5 + 3] = boxes[(offset + threadIdx.x)*5 + 3] ;
        blockBoxes[threadIdx.x*5 + 4] = boxes[(offset + threadIdx.x)*5 + 4] ;
    }

    // ensure that all threads in the block will have access to all boxes 
    // assigned to that block   
    __syncthreads() ;

    // process
    if (threadIdx.x < numRows)
    {
        const int boxIdx = BLOCKSIZE * rowIdx + threadIdx.x ;
        const T *currBox = boxes + boxIdx * 5 ;

        // use a bit mask to store box overlaps above the threshold
        unsigned long long tt = 0 ;

        // if current block lies on the diagonal of the grid, apply offset
        // (thi is to prevent a box from later being removed for having 
        // overlap with itself)
        int start = 0 ;
        if (rowIdx == colIdx) start = threadIdx.x + 1 ;

        // compare the current box against every other box in its block and
        // track its index if its overlap exceeds the threshold
        for (int ii = start; ii < numCols; ii++)
        {
            if (jaccard(currBox, blockBoxes + ii*5) > overlapThresh)
            {
                tt |= 1ULL << ii ;
            }
        }
        mask[boxIdx * colBlocks + colIdx] = tt ;
    }
}



namespace vl { namespace impl {

    template<typename T>
    struct bboxnms<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context,
            std::vector<int> &output,
            T const* boxes,
            float overlapThresh, 
            size_t numBoxes,
            int &numKept) 
    {
    const int colBlocks = DIVROUNDUP(numBoxes, BLOCKSIZE);

    // Allocate memory to hold the nms results mask 
    unsigned long long *mask = NULL;
    int MASK_ARRAY_BYTES = sizeof(unsigned long long) * numBoxes * colBlocks ;
    hipMalloc(&mask, MASK_ARRAY_BYTES);
    
    // we will only use the x-dim on each thread block
    dim3 threads(BLOCKSIZE) ; 

    // the thread blocks are arranged as an square grid
    dim3 blocks(DIVROUNDUP(numBoxes, BLOCKSIZE), DIVROUNDUP(numBoxes, BLOCKSIZE)) ;

    nmsKernel<T><<<blocks,threads>>>(numBoxes, overlapThresh, boxes, mask, colBlocks) ;

    // use mask_h to hold results and copy back from device
    std::vector<unsigned long long> mask_h(numBoxes * colBlocks);
    hipMemcpy(&mask_h[0], mask, MASK_ARRAY_BYTES, hipMemcpyDeviceToHost);

    // use `remv` to keep track of which blocks have been processed 
    std::vector<unsigned long long> remv(colBlocks);
    memset(&remv[0], 0, sizeof(unsigned long long) * colBlocks);

    for (int ii = 0; ii < numBoxes; ii++)
    {
        int blockNum = ii / BLOCKSIZE ;
        int inblock = ii % BLOCKSIZE ;

        // check that the current box has not yet been "removed"
        if (!(remv[blockNum] & (1ULL << inblock)))
        {
            output[numKept] = ii ;  // store box index
            numKept += 1 ;
            unsigned long long *p = &mask_h[0] + ii * colBlocks ;

            // remove boxes with high overlap following the current one
            for (int jj = blockNum; jj < colBlocks; jj++)
            {
                remv[jj] |= p[jj] ;
            }
        }
    }
    hipFree(mask);  

    return VLE_Success ;
   }
 } ;
} } // namespace vl::impl

template struct vl::impl::bboxnms<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::bboxnms<vl::VLDT_GPU, double> ;
#endif
